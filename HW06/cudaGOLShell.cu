/*THIS IS SHELL CODE
compile with nvcc -Xcompiler -fopenmp -o cudaGOLShell cudaGOLShell.cu  -arch=sm_20
run with ./cudaGOLShell boardfile.txt
NOTE: fixed printing for non-square boards.
*/
// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>
/*
Put #define statements here 
*/
#define Treduction 256

// function prototypes
void printBoard(int, int, int* );
void updateBoard(int, int, int*, int*);

/*Q1: put your cudaUpdateBoard kernel here*/
__global__ void cudaUpdateBoard(int N, int M,  int* oldBoard, int* newBoard){
  
  
  // Declare variables
  int i, j;
  int cell, cellBelow, cellAbove;
  int sum, sumBelow, sumLevel, sumAbove;
  int oldState, newState;
  
  for( i=1;i<N+1;++i){  //starting at 1 to skip boundary layer
    for( j=1;j<M+1;++j){ //starting at 1 to skip boundary layer
      
      // Make useful indices
      cell = j+i*(M+2); // Current cell
      cellBelow = j+(i-1)*(M+2); // Cell below it
      cellAbove = j+(i+1)*(M+2); // Cell above it
      
      // Split the sum into the 3 above, 3 below, and 2 level neighbors
      sumBelow = oldBoard[cellBelow]+oldBoard[cellBelow-1]+oldBoard[cellBelow+1];
      sumLevel = oldBoard[cell-1]+oldBoard[cell+1];
      sumAbove = oldBoard[cellAbove]+oldBoard[cellAbove-1]+oldBoard[cellAbove+1];
      
      // Compute the sum
      sum = sumBelow + sumLevel + sumAbove;
      
      
      // Get the current state of the cell
      oldState = oldBoard[cell];
      
      // Game of life rules:
      // If the cell was alive
      if(oldState == 1){
        
        // Exactly 2 or 3 neighbors
        if(sum == 2||sum == 3){
          newState = 1;
        }
        
        // More then 3 or less then 2 neighbors
        else{
          newState = 0;
        }
      }
      
      // If the cell was dead
      else{
        
        // Exactly 3 neighbors
        if(sum == 3){
          newState = 1;
        }
        
        else{
          newState = 0;
        }
      }
      
      // Update new board
      newBoard[cell] = newState;
      
      //
    }
  } 
}


/*Q2: put your cudaUpdateChecker kernel here*/
__global__ void cudaUpdateChecker(int N, int M, int* newBoard, int* oldBoard, int* partialSums){
  int t = threadIdx.x;
  int b = blockIdx.x;
  int id = t + b*Treduction;
  __shared__ int sumBlock[Treduction];
  sumBlock[t] = 0;
  __syncthreads();
  if (id < M*N && id >= 0){
     if (newBoard[id] - oldBoard[id] != 0)
     	sumBlock[t] = 1;
  }
    int alive = Treduction/2;
    while(alive >= 1){
    		__syncthreads();
		if (t < alive) sumBlock[t] += sumBlock[t+alive];
		alive /= 2;
    }
    partialSums[b] = sumBlock[0];
}



// main
int main(int argc, char **argv){
  
  // Board dimensions
  int N, M;
  
  // Read input file containing board information and number of iterations
  FILE *fp = fopen(argv[1], "r");
  
  if(fp==NULL){
    printf("Game Of Life: could not load input file %s\n", argv[1]);
    exit(0);
  }
  // keep reading the file until you find $Size
  char buf[BUFSIZ];
  do{
    fgets(buf, BUFSIZ, fp);
  }while(!strstr(buf, "$Size"));
  
  // read the size
  fgets(buf, BUFSIZ, fp);
  sscanf(buf, "%d %d", &N, &M);
  
  // Initialize boards
  int *boardA = (int*) calloc((N+2)*(M+2), sizeof(int));
  int *boardB = (int*) calloc((N+2)*(M+2), sizeof(int));
  int ii;
  /* DO NOT REMOVE THIS PART */
  for (ii=0; ii<(N+2)*(M+2); ii++){
    boardA[ii] = 0;
    boardB[ii] = 0;
  }
  
  // Read number of updates
  int T;
  do{
    fgets(buf, BUFSIZ, fp);
  }while(!strstr(buf, "$Updates"));
  fgets(buf, BUFSIZ, fp);
  sscanf(buf, "%d", &T);
  
  printf("number of updates: %d\n", T);
  int numAlive;
  
  // next, scan for how many alive cells you have
  do{
    fgets(buf, BUFSIZ, fp);
  }while(!strstr(buf, "$Alive"));
  
  // read the number of alive cells
  fgets(buf, BUFSIZ, fp);
  sscanf(buf, "%d", &numAlive);
  printf("initial number of alive cells:  %d \n", numAlive);
  
  //allocate the alive list (one list per every dimension
  int * LiveList_i = (int*) calloc(numAlive, sizeof(int));
  int * LiveList_j = (int*) calloc(numAlive, sizeof(int));
  
  for (int i=0; i<numAlive; i++){
    fgets(buf, BUFSIZ, fp);
    sscanf(buf, "%d %d", &LiveList_i[i], &LiveList_j[i]);
  }
  
  fclose(fp);
  // Spawn Cells
  for(int n = 0; n<numAlive; ++n){
    int i = LiveList_i[n]; int j = LiveList_j[n];
    boardA[j+i*(M+2)] = 1;
  }
  free(LiveList_i);
  free(LiveList_j);
  
  // Print Initial Board
  printf("Initial Condition\n");
  if ((N<=60) &&(M<=60)){
    printBoard(N,M,boardA);
  }

    
  // Start Game
  int K =M*N;
  /*Q1: create and allocate DEVICE boards A and B here */
  int *DboardA, *DboardB; 
  hipMalloc(&DboardA, K*sizeof(int));
  hipMalloc(&DboardB, K*sizeof(int));
  /*Q1: copy boardA to DEVICE boards A here */
  hipMemcpy(DboardA, boardA, K*sizeof(int), hipMemcpyHostToDevice);
  
  
  int t = 0;
  int changes;
 
 /*Q1: set the number of blocks and threads here*/
  dim3 grid(4,4);
  dim3 block(8,8);

  /*Q2: set the number of blocks and threads here*/
  int numblocks = K/Treduction;
  int numthreads = Treduction;
  
  /*Q2: create and allocate DEVICE boards for partial sums */
  int *Dsum;
  hipMalloc(&Dsum, numblocks*sizeof(int));

  /*Q2: create and allocate HOST board for partial sums */
  int *Hsum = (int*) calloc(numblocks, sizeof(int));
 
  // we time using OpenMP timing functions
  double t1, t2;
// REMEMBER TO TURN OFF ALL DISPLAY COMMANDS BEFORE TIMING!!! 
t1= omp_get_wtime();

  while(t<T){
    
    // Update boardA into boardB
    /* Q1: replace this call with cudaUpdateBoard*/
    cudaUpdateBoard <<< grid, block >>> (N, M, DboardA, DboardB);
 
    // check for changes
    /* Q2: call cudaUpdateChecker here*/
    cudaUpdateChecker <<< numblocks, numthreads >>> (N, M, DboardB, DboardA, Dsum);

   /* Q2: copy the partial DEVICE sum array to host here*/
    hipMemcpy(Hsum,Dsum,numblocks*sizeof(int), hipMemcpyDeviceToHost);

    changes  = 0;
   /* Q2: sum the entries of the partial sum array, check for if still life*/
    for (int i = 0;i<numblocks;++i){
    	changes += Hsum[i];
}

    hipMemcpy(boardB,DboardB,K*sizeof(int), hipMemcpyDeviceToHost);
    if (changes == 0){
       printf("Board has reached still-life\n");
       break;
     }

    //copy for display
   /* Q2: copy DEVICE board A to HOST boardA for display*/
    hipMemcpy(boardA, DboardA, K*sizeof(int), hipMemcpyDeviceToHost);

    // display if the board is small
    ++t;
    printf("updated, t = %d\n", t);
    if ((N<=60) &&(M<=60)){
       printBoard(N,M,boardB);
    }
    if(t==T) break;
   
    // Update boardB into boardA
    /* Q1: replace this call with cudaUpdateBoard*/
    cudaUpdateBoard <<< grid, block >>> (N, M, DboardB, DboardA);

    //check for changes
   /* Q2: call cudaUpdateChecker here*/
   cudaUpdateChecker <<< numblocks, numthreads >>> (N, M, DboardA, DboardB, Dsum);
 
   /* Q2: copy the partial DEVICE sum array to host here*/
   hipMemcpy(Hsum, Dsum, numblocks*sizeof(int), hipMemcpyDeviceToHost);
    
  /* Q2: sum the entries of the partial sum array, check for if still life*/
    changes  = 0;
    for (int i = 0;i<numblocks;++i)
    	changes += Hsum[i];

    hipMemcpy(boardA, DboardA, K*sizeof(int), hipMemcpyDeviceToHost);
    if (changes == 0){
       printf("Board has reached still-life\n");
       break;
    }
  
    //copy board for display
    /* Q2: copy DEVICE board A to HOST boardA for display*/

    //display board if small enough
    ++t;
    printf("updated, t = %d \n", t);
    if ((N<=60) &&(M<=60)){
      printBoard(N,M,boardA);
    }

    if(t==T) break;
    //check for still-life
    
  }
 t2 = omp_get_wtime();
printf("it took %f seconds\n", t2-t1);  
// Finish
  free(boardA);
  free(boardB);
  
  //free cuda variables
  /*Q1 and Q2: free DEVICE variables using cudaFree*/
  hipFree(DboardA);
  hipFree(DboardB);
  hipFree(Dsum);
  hipFree(Hsum);

  return(0);
}

void updateBoard(int N, int M,  int* oldBoard, int* newBoard){
  
  // Declare variables
  int i, j;
  int cell, cellBelow, cellAbove;
  int sum, sumBelow, sumLevel, sumAbove;
  int oldState, newState;
  
  for( i=1;i<N+1;++i){  //starting at 1 to skip boundary layer
    for( j=1;j<M+1;++j){ //starting at 1 to skip boundary layer
      
      // Make useful indices
      cell = j+i*(M+2); // Current cell
      cellBelow = j+(i-1)*(M+2); // Cell below it
      cellAbove = j+(i+1)*(M+2); // Cell above it
      
      // Split the sum into the 3 above, 3 below, and 2 level neighbors
      sumBelow = oldBoard[cellBelow]+oldBoard[cellBelow-1]+oldBoard[cellBelow+1];
      sumLevel = oldBoard[cell-1]+oldBoard[cell+1];
      sumAbove = oldBoard[cellAbove]+oldBoard[cellAbove-1]+oldBoard[cellAbove+1];
      
      // Compute the sum
      sum = sumBelow + sumLevel + sumAbove;
      
      
      // Get the current state of the cell
      oldState = oldBoard[cell];
      
      // Game of life rules:
      // If the cell was alive
      if(oldState == 1){
        
        // Exactly 2 or 3 neighbors
        if(sum == 2||sum == 3){
          newState = 1;
        }
        
        // More then 3 or less then 2 neighbors
        else{
          newState = 0;
        }
      }
      
      // If the cell was dead
      else{
        
        // Exactly 3 neighbors
        if(sum == 3){
          newState = 1;
        }
        
        else{
          newState = 0;
        }
      }
      
      // Update new board
      newBoard[cell] = newState;
      
      //
    }
  }
  
}


void printBoard(int N, int M, int* board){
  
  int i, j, cell, state;
  //Formatted to start in top left corner, moving across each row
  for(i=1;i<N+1;++i){
    for(j=1;j<M+1;++j){ //starting at 1 to skip boundary layer
      
      // Cell number and state
    
      cell = j + i*(M+2);
   //   printf("i= %d j = %d this is cell %d \n",i,j, cell );
      state = board[cell];
      
      if(state == 1){
        printf("X ");
      }
      
      else{
        printf(". ");
      }
      
    }
    printf("\n");
  }
}
